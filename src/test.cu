#include <iostream>
#include <time.h>
#include <unistd.h>
#include "Int.cuh"

using namespace mmath;
int main() {
	// 計測用(start)
	float elapsed;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	Int c("deadbeefabcdef1234567890");
	std::cout << c << std::endl;

	// 計測用(end)
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	std::cout << "elapsed: " << elapsed << " [ms]" << std::endl;
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
