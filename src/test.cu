#include <iostream>
#include <time.h>
#include <unistd.h>
#include "Int.hpp"

using namespace mmath;
int main() {
	// 計測用
	float elapsed;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	Digits d(1ul << 10, 50);

	hipEventRecord(start, 0);

	d.to_zero();
	d.print();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	std::cout << "elapsed: " << elapsed << " [ms]" << std::endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
