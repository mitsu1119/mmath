#include <iostream>
#include <time.h>
#include <unistd.h>
#include "Int.hpp"

using namespace mmath;
int main() {
	// 計測用
	float elapsed;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	Int a("1234");

	hipEventRecord(start, 0);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	std::cout << "elapsed: " << elapsed << " [ms]" << std::endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
