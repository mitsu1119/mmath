#include <iostream>
#include <time.h>
#include <unistd.h>
#include "Int.cuh"

using namespace mmath;
int main() {
	// 計測用
	float elapsed;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	Int a("123456789");
	Int b("ab239438902");
	Int c("aaaaaaaaaa");

	hipEventRecord(start, 0);


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	std::cout << "elapsed: " << elapsed << " [ms]" << std::endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
