#include <iostream>
#include <time.h>
#include <unistd.h>
#include "Int.cuh"

using namespace mmath;
int main() {

	Int c("ffffffffabcdef123456789ffffffffffabcdef1234567890abcdef1234567890deadbeefdeadbeefabcdef1234567890abcdef1234567890deadbeefdeadbeefabcdef1234567890");
	Int d("ffffffffabcdef123456789ffffffffffabcdef1234567890abcdef1234567890deadbeefdeadbeefabcdef1234567890abcdef1234567890deadbeefdeadbeefabcdef1234567890");

	// 計測用(variable)
	float elapsed;
	hipEvent_t start, stop;

	// 計測用(start)
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	c.mul(d);

	// 計測用(end)
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	std::cout << "elapsed: " << elapsed << " [ms]" << std::endl;
	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout << c << std::endl;

	return 0;
}
